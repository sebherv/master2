
#include <hip/hip_runtime.h>
#include <cstdio>

void VecAdd( const float* pA, const float* pB, float* pC, int vectorSize)
{
	for( int i = 0 ; i < vectorSize; ++i)
	{
		pC[i] = pA[i] + pB[i];
	}
}

void VecFill( float * pVector, int vectorSize, float firstValue, float increment )
{
	for( int i = 0 ; i < vectorSize; ++i)
	{
		pVector[i] = firstValue+(increment*i);
	}
}

void VecPrint( const float * pVector, int vectorSize, const char * pLabel="noname" )
{
	printf("%s : \n", pLabel);
	for( int i = 0 ; i < vectorSize; ++i)
	{
		printf("[%d] : %f\n", i, pVector[i]);
	}
}

int main(int argc, char* argv[])
{
	int vectorSize = 10;
	
	size_t vectorMemSize = vectorSize * sizeof(float);
	
	float *pA = (float *)malloc(vectorMemSize);
	float *pB = (float *)malloc(vectorMemSize);
	float *pC = (float *)malloc(vectorMemSize);
	
	VecFill(pA, vectorSize, 0.0f, 1.0f);
	VecPrint(pA, vectorSize, "A");
	VecFill(pB, vectorSize, 10.0f, -0.5f);
	VecPrint(pB, vectorSize, "B");
	VecAdd(pA, pB, pC, vectorSize);
	VecPrint(pC, vectorSize, "A+B");
	
	free(pA);
	free(pB);
	free(pC);

	return 0;
}

