
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void VecAddKernel(float* dA, float* dB, float* dC)
{
	int i = threadIdx.x;
	dC[i] = dA[i] + dB[i];
}

void VecAdd( const float* pA, const float* pB, float *pC, int vectorSize)
{
	size_t vectorMemSize = vectorSize * sizeof(float);
	// Allocate
	float* dA;
	hipMalloc(	&dA, vectorMemSize );
	float* dB;
	hipMalloc(	&dB, vectorMemSize );
	float* dC;
	hipMalloc(	&dC, vectorMemSize );
	//vectors in device memory
	// copy vectors from host memory to device memory
	hipMemcpy( dA, pA, vectorMemSize, hipMemcpyHostToDevice);
	hipMemcpy( dB, pB, vectorMemSize, hipMemcpyHostToDevice);
	dim3 numBlocks(1,1,1);
	dim3 numThreadsPerBlock(vectorSize,1,1);
	// Kernel invocation
	VecAddKernel<<<numBlocks, numThreadsPerBlock>>>(dA, dB, dC);
	// copy the resulting vector from device memory to host memory
	hipMemcpy( pC, dC, vectorMemSize, hipMemcpyDeviceToHost);
	// free device memory
	hipFree( dA );
	hipFree( dB );
	hipFree( dC );
}

void VecFill( float * pVector, int vectorSize, float firstValue, float increment )
{
	for( int i = 0 ; i < vectorSize; ++i)
	{
		pVector[i] = firstValue+(increment*i);
	}
}

void VecPrint( const float * pVector, int vectorSize, const char * pLabel="noname" )
{
	printf("%s : \n", pLabel);
	for( int i = 0 ; i < vectorSize; ++i)
	{
		printf("[%d] : %f\n", i, pVector[i]);
	}
}

int main(int argc, char* argv[])
{
	int vectorSize = 10;
	
	size_t vectorMemSize = vectorSize * sizeof(float);
	
	float *pA = (float *)malloc(vectorMemSize);
	float *pB = (float *)malloc(vectorMemSize);
	float *pC = (float *)malloc(vectorMemSize);
	
	VecFill(pA, vectorSize, 0.0f, 1.0f);
	VecPrint(pA, vectorSize, "A");
	VecFill(pB, vectorSize, 10.0f, -0.5f);
	VecPrint(pB, vectorSize, "B");
	VecAdd(pA, pB, pC, vectorSize);
	VecPrint(pC, vectorSize, "A+B");
	
	free(pA);
	free(pB);
	free(pC);

	return 0;
}

